#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "sha1_.cu"
#include <string.h>

#define N 8
#define SPACE 10000
#define BLOCK_SIZE 16   





__global__ void kernel(unsigned char* digest, unsigned char* find, bool* bingo) {

	// keep the context in shared memory
	__shared__ unsigned char ctx[16][16][N];

	// keep the digest in the shared memory, too
	//__shared__ unsigned char target[20];
	
	// the digest we calculate 
	__shared__ unsigned char result[16][16][20];
	
	// 00 - 99
	int high = blockIdx.x * blockDim.x + threadIdx.x; if (high >= 10000) return;
	// 00- 99 
	int low  = blockIdx.y * blockDim.y + threadIdx.y; if (low >= 10000) return;
	
	/*
	// only one thread of a block has the responsibility to dump the digest
	if (threadIdx.x==0 && threadIdx.y==0) {
		for (int i=0; i<20; i++)
			target[i] = digest[i];
	}
	__syncthreads(); // !!
	*/

	// generate and assign context
	ctx[threadIdx.x][threadIdx.y][0] = (unsigned char)(high /1000 + 48);
	ctx[threadIdx.x][threadIdx.y][1] = (unsigned char)((high % 1000) / 100 + 48);
	ctx[threadIdx.x][threadIdx.y][2] = (unsigned char)((high % 100) / 10 + 48);
	ctx[threadIdx.x][threadIdx.y][3] = (unsigned char)(high % 10 + 48);
	ctx[threadIdx.x][threadIdx.y][4] = (unsigned char)(low/1000 + 48);
	ctx[threadIdx.x][threadIdx.y][5] = (unsigned char)((low % 1000) / 100 + 48);
	ctx[threadIdx.x][threadIdx.y][6] = (unsigned char)((low % 100) / 10 + 48);
	ctx[threadIdx.x][threadIdx.y][7] = (unsigned char)((low % 10 + 48));
	
	// sha1
	sha1(result[threadIdx.x][threadIdx.y], ctx[threadIdx.x][threadIdx.y], N);
	
	// compare the result to the digest 
	int flag = 1;
	for (int i=0; i<20; i++) {
		if (result[threadIdx.x][threadIdx.y][i] != digest[i]) {
			flag = 0;
			break;
		}
	}


	//find !!
	if (flag==1) {
		find[0] = ctx[threadIdx.x][threadIdx.y][0];
		find[1] = ctx[threadIdx.x][threadIdx.y][1];
		find[2] = ctx[threadIdx.x][threadIdx.y][2];
		find[3] = ctx[threadIdx.x][threadIdx.y][3];
		find[4] = ctx[threadIdx.x][threadIdx.y][4];
		find[5] = ctx[threadIdx.x][threadIdx.y][5];
		find[6] = ctx[threadIdx.x][threadIdx.y][6];
		find[7] = ctx[threadIdx.x][threadIdx.y][7];
		*bingo = true;
	}
}


int main(int argc, char** argv) {
    

	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (argc <1) {
		printf("wrong arguments\n");
		return -1;
	}


	// readin
    char* input = argv[1];


	unsigned char cypher[20];
    
	for (int i=0; i<20;i++) {
        unsigned char high = input[2*i];
        unsigned char low = input[2*i+1];
		unsigned char combine;
        switch(high) {
case '1': combine = 0x10; break;
case '2': combine = 0x20; break;
case '3': combine = 0x30; break;
case '4': combine = 0x40; break;		  
case '5': combine = 0x50; break;
case '6': combine = 0x60; break;
case '7': combine = 0x70; break;
case '8': combine = 0x80; break;
case '9': combine = 0x90; break;
case 'a': combine = 0xa0; break;
case 'b': combine = 0xb0; break;
case 'c': combine = 0xc0; break;
case 'd': combine = 0xd0; break;
case 'e': combine = 0xe0; break;
case 'f': combine = 0xf0; break;
default: combine = 0x00;
		}
		switch(low) {
case '1': combine |= 0x01; break;
case '2': combine |= 0x02; break;
case '3': combine |= 0x03; break;
case '4': combine |= 0x04; break;		  
case '5': combine |= 0x05; break;
case '6': combine |= 0x06; break;
case '7': combine |= 0x07; break;
case '8': combine |= 0x08; break;
case '9': combine |= 0x09; break;
case 'a': combine |= 0x0a; break;
case 'b': combine |= 0x0b; break;
case 'c': combine |= 0x0c; break;
case 'd': combine |= 0x0d; break;
case 'e': combine |= 0x0e; break;
case 'f': combine |= 0x0f; break;
default: combine |= 0x00;
		}
		cypher[i] = combine;
    }

	printf("\ncypher:");
	for (int i=0; i<20; i++) {
		printf("%x", cypher[i]);
	}
	printf("\n");


	// cypher has been prepared
    unsigned char *digest, *digest_d, *find_d, *find;
	
	digest = cypher;

	dim3 blocksPerGrid((10000+15)/16, (10000+15)/16);
	dim3 threadsPerBlock(16, 16);

	// digest
	hipMalloc((void**) &digest_d, sizeof(unsigned char)*20);
	hipMemcpy(digest_d, digest, sizeof(unsigned char)*20, hipMemcpyHostToDevice);

	// find output
	hipMalloc((void**) &find_d, sizeof(unsigned char)*N);
	find = (unsigned char*) malloc(sizeof(unsigned char)*N);

	// bingo
	bool *bingo, *bingo_d;
	bingo = (bool *) malloc(sizeof(bool));
	*bingo = false;
	hipMalloc((void**) &bingo_d, sizeof(bool));
	hipMemcpy(bingo_d, bingo, sizeof(bool), hipMemcpyHostToDevice);
	

	hipEventRecord(start, 0);
	kernel<<<blocksPerGrid, threadsPerBlock>>>(digest_d, find_d, bingo_d);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	// get the output
	hipMemcpy(find, find_d, sizeof(unsigned char)*N, hipMemcpyDeviceToHost);
	hipMemcpy(bingo, bingo_d, sizeof(bool), hipMemcpyDeviceToHost);

	if (*bingo==true) {
		printf("\nbingo!\n");

		printf("\nplain:");
		for (int i=0; i<N; i++)
			printf("%c", find[i]);
	} else {
		printf("not found!");
	}
	printf("\ntime:%f\n", elapsedTime);
	

	hipFree(find_d);
	hipFree(digest_d);
	hipFree(bingo_d);
	free(find);
	free(bingo);
	return 0;
}
